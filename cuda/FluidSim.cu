#include "hip/hip_runtime.h"
#include "FluidSim.h"
#include <hip/hip_runtime.h>

#define HANDLE_CUDA_CALL(cuda_func, err) \
do {\
	hipError_t err_code;\
	if((err_code = cuda_func) != hipSuccess) { \
		cerr << "Cuda error in call: " << err  << " with error code: " << err_code << endl; \
		exit(1); \
	}\
} while(0)

// Computation depends on neighboring results, thus it can't be parallelized
void FluidSim::solvePoisson() {
	double dx2 = m_dx * m_dx;
	double residual = m_acc + 1; // initial residual
	double rho = 1;

	Array2d& p = p_pressure->x();

	for (int it = 0; residual > m_acc && it < m_iter; ++it) {
		// Note that the boundaries are handles by the framework, so you iterations should be similar to:
		for (int y = 1; y < m_res_y - 1; ++y) {
			for (int x = 1; x < m_res_x - 1; ++x) {
				double b = -p_divergence->x()(x, y) / m_dt * rho; // right-hand
				// TODO: update the pressure values
				p(x, y) = (dx2 * b +
					p(x - 1, y) + p(x + 1, y) +
					p(x, y - 1) + p(x, y + 1)) / 4.0;
				
			}
		}

		// Compute the new residual, i.e. the sum of the squares of the individual residuals (squared L2-norm)
		residual = 0;
		for (int y = 1; y < m_res_y - 1; ++y) {
			for (int x = 1; x < m_res_x - 1; ++x) {
				double b = -p_divergence->x()(x, y) / m_dt * rho; // right-hand
				// TODO: compute the cell residual
				double cellResidual = b - (4 * p(x, y) -
					p(x - 1, y) - p(x + 1, y) -
					p(x, y - 1) - p(x, y + 1)) / dx2;

				residual += cellResidual * cellResidual;

			}
		}

		// Get the L2-norm of the residual
		residual = sqrt(residual);

		// We assume the accuracy is meant for the average L2-norm per grid cell
		residual /= (m_res_x - 2) * (m_res_y - 2);

		//// For your debugging, and ours, please add these prints after every iteration
		//cout << "Pressure solver: iter=" << it << ", res=" << residual << endl;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////CUDA CORRECT_VELOCITY///////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void correctVelocity_kernel_u(double* cu_p, double* cu_u, double m_dt_idx, int warpsPerBlock, int length) {
	// Compute current thread index
	int j = blockIdx.x + 1; // +1 accounts for the array indexing starting at 1
	int i = threadIdx.x * warpsPerBlock + 1; 

	int array_idx = i * length + j;

	// Loop for thread coalescing
	for (int l = 0; l < warpsPerBlock; ++l) {
		cu_u[array_idx] = cu_u[array_idx] - (m_dt_idx * (cu_p[array_idx]- cu_p[(i-1) * length + j]));
	}
}

__global__ void correctVelocity_kernel_v(double* cu_p, double* cu_v, double m_dt_idx, int warpsPerBlock, int length) {
	// Compute current thread index
	int j = blockIdx.x + 1; // +1 accounts for the array indexing starting at 1
	int i = threadIdx.x * warpsPerBlock + 1;

	int array_idx = i * length + j;

	// Loop for thread coalescing
	for (int l = 0; l < warpsPerBlock; ++l) {
		cu_v[array_idx] = cu_v[array_idx] - (m_dt_idx * (cu_p[array_idx] - cu_p[i * length + j-1]));
	}
}

// CUDA Correct Velocity
void FluidSim::correctVelocity() {
	Array2d& p = p_pressure->x();
	Array2d& u = p_velocity->x();
	Array2d& v = p_velocity->y();

	// Precompute for kernels
	double m_dt_idx = m_dt * m_idx;

	// Retrieve device property
	int max_thrds = prop.maxThreadsPerBlock;

	// Compute n_iterations for both u & v
	int length_u = (m_res_y - 1) * m_res_x;
	int length_v = m_res_y * (m_res_x - 1);

	// Compute CUDA-specific parallelization parameters
	int nBlks_u = length_u - 2;
	int nBlks_v = length_v - 2;

	// Clamp the number of threads to the maximum amount allowed by the device
	int threadsPerBlk_u = fmin(nBlks_u, max_thrds);
	int threadsPerBlk_v = fmin(nBlks_v, max_thrds);

	// Compute the number of threads per block
	int warpsPerBlock_u = ((nBlks_u * threadsPerBlk_u) + max_thrds - 1) / max_thrds;
	int warpsPerBlock_v = ((nBlks_v * threadsPerBlk_v) + max_thrds - 1) / max_thrds;

	// Prepare GPU variables
	double *cu_p, *cu_u, *cu_v;
	double* p_addr = &(p(0,0));
	double* u_addr = &(u(0,0));
	double* v_addr = &(v(0,0));

	int size_p = p.size(0) * p.size(1) * sizeof(double);
	int size_u = u.size(0) * u.size(1) * sizeof(double);
	int size_v = v.size(0) * v.size(1) * sizeof(double);

	// Try to allocate space for p, u, and v on GPU
	HANDLE_CUDA_CALL(hipMalloc((void**)&cu_p, size_p), "Malloc p");
	HANDLE_CUDA_CALL(hipMalloc((void**)&cu_u, size_u), "Malloc u");
	HANDLE_CUDA_CALL(hipMalloc((void**)&cu_v, size_v), "Malloc v");

	// Copy current array data from host to device
	HANDLE_CUDA_CALL(hipMemcpy(cu_p, p_addr, size_p, hipMemcpyHostToDevice), "Cpy p to device");
	HANDLE_CUDA_CALL(hipMemcpy(cu_u, u_addr, size_u, hipMemcpyHostToDevice), "Cpy u to device");
	HANDLE_CUDA_CALL(hipMemcpy(cu_v, v_addr, size_v, hipMemcpyHostToDevice), "Cpy v to device");

	// Call kernels
	correctVelocity_kernel_u<<<nBlks_u, threadsPerBlk_u>>>(cu_p, cu_u, m_dt_idx, warpsPerBlock_u, length_u);
	correctVelocity_kernel_v<<<nBlks_v, threadsPerBlk_v>>>(cu_p, cu_v, m_dt_idx, warpsPerBlock_v, length_v);

	// Copy results back from the device to the host (p isn't modified so no need)
	HANDLE_CUDA_CALL(hipMemcpy(u_addr, cu_u, size_u, hipMemcpyDeviceToHost), "Cpy u to Host");
	HANDLE_CUDA_CALL(hipMemcpy(v_addr, cu_v, size_v, hipMemcpyDeviceToHost), "Cpy v to Host");

	// Free CUDA variables
	hipFree(cu_p);
	hipFree(cu_u);
	hipFree(cu_v);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////END: CUDA CORRECT_VELOCITY////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void FluidSim::advectValues() {
	// store original values
	Array2d d(p_density->x());
	Array2d u(p_velocity->x());
	Array2d v(p_velocity->y());
	// move forward
	advectDensitySL(u, v);
	advectVelocitySL(u, v);

	if (m_macOn){
		// store forward advection
		Array2d d_forward(p_density->x());
		Array2d u_forward(p_velocity->x());
		Array2d v_forward(p_velocity->y());
		MacCormackUpdate(d, d_forward, u, u_forward, v, v_forward);
		MacCormackClamp(d, d_forward, u, u_forward, v, v_forward);
	}
}

void FluidSim::advectDensitySL(const Array2d& u, const Array2d& v) {
	Array2d& d = p_density->x();
	Array2d d_tmp(d.size(0), d.size(1));

	// Densities, grid centers
	for (int y = 1; y < m_res_y - 1; ++y) {
		for (int x = 1; x < m_res_x - 1; ++x) {
			// TODO: Compute the velocity
			double last_x_velocity = (u(x, y) + u(x + 1, y)) / 2;
			double last_y_velocity = (v(x, y) + v(x, y + 1)) / 2;

			// TODO: Find the last position of the particle (in grid coordinates)
			double last_x = x - m_dt * m_idx * last_x_velocity;
			double last_y = y - m_dt * m_idx * last_y_velocity;

			// Make sure the coordinates are inside the boundaries
			// Densities are known between 1 and res-2
			if (last_x < 1) last_x = 1;
			if (last_y < 1) last_y = 1;
			if (last_x > m_res_x - 2) last_x = m_res_x - 2;
			if (last_y > m_res_y - 2) last_y = m_res_y - 2;

			// Determine corners for bilinear interpolation
			int x_low = (int)last_x;
			int y_low = (int)last_y;
			int x_high = x_low + 1;
			int y_high = y_low + 1;

			// Compute the interpolation weights
			double x_weight = last_x - x_low;
			double y_weight = last_y - y_low;

			// TODO: Bilinear interpolation
			d_tmp(x, y) = x_weight * y_weight * d(x_high, y_high) +
				(1 - x_weight) * y_weight * d(x_low, y_high) +
				x_weight * (1 - y_weight) * d(x_high, y_low) +
				(1 - x_weight) * (1 - y_weight) * d(x_low, y_low);

		}
	}

	// Copy the values in temp to the original buffers
	d = d_tmp;
}

void FluidSim::advectVelocitySL(const Array2d& u, const Array2d& v) {
	Array2d& u_in = p_velocity->x();
	Array2d& v_in = p_velocity->y();

	Array2d u_tmp(u_in.size(0), u_in.size(1));
	Array2d v_tmp(v_in.size(0), v_in.size(1));

	// Velocities (u), MAC grid
	for (int y = 1; y < m_res_y - 1; ++y) {
		for (int x = 1; x < m_res_x; ++x) {
			// TODO: Compute the velocity
			double last_x_velocity = u(x, y);
			double last_y_velocity = (v(x, y) + v(x - 1, y) + v(x - 1, y + 1) + v(x, y + 1)) / 4;

			// TODO: Find the last position of the particle (in grid coordinates)
			double last_x = x - m_dt * m_idx * last_x_velocity;
			double last_y = y - m_dt * m_idx * last_y_velocity;

			// Make sure the coordinates are inside the boundaries
			// Being conservative, one can say that the velocities are known between 1.5 and res-2.5
			// (the MAC grid is inside the known densities, which are between 1 and res - 2)
			if (last_x < 1.5) last_x = 1.5;
			if (last_y < 1.5) last_y = 1.5;
			if (last_x > m_res_x - 1.5) last_x = m_res_x - 1.5;
			if (last_y > m_res_y - 2.5) last_y = m_res_y - 2.5;

			// Determine corners for bilinear interpolation
			int x_low = (int)last_x;
			int y_low = (int)last_y;
			int x_high = x_low + 1;
			int y_high = y_low + 1;

			// Compute the interpolation weights
			double x_weight = last_x - x_low;
			double y_weight = last_y - y_low;

			// TODO: Bilinear interpolation
			u_tmp(x, y) = x_weight * y_weight * u_in(x_high, y_high) +
				(1 - x_weight) * y_weight * u_in(x_low, y_high) +
				x_weight * (1 - y_weight) * u_in(x_high, y_low) +
				(1 - x_weight) * (1 - y_weight) * u_in(x_low, y_low);

		}
	}

	// Velocities (v), MAC grid
	for (int y = 1; y < m_res_y; ++y) {
		for (int x = 1; x < m_res_x - 1; ++x) {
			// TODO: Compute the velocity
			double last_x_velocity = (u(x, y) + u(x + 1, y) + u(x + 1, y - 1) + u(x, y - 1)) / 4;
			double last_y_velocity = v(x, y);

			// TODO: Find the last position of the particle (in grid coordinates)
			double last_x = x - m_dt * m_idx * last_x_velocity;
			double last_y = y - m_dt * m_idx * last_y_velocity;

			// Make sure the coordinates are inside the boundaries
			// Being conservative, one can say that the velocities are known between 1.5 and res-2.5
			// (the MAC grid is inside the known densities, which are between 1 and res - 2)
			if (last_x < 1.5) last_x = 1.5;
			if (last_y < 1.5) last_y = 1.5;
			if (last_x > m_res_x - 2.5) last_x = m_res_x - 2.5;
			if (last_y > m_res_y - 1.5) last_y = m_res_y - 1.5;

			// Determine corners for bilinear interpolation
			double x_low = (int)last_x;
			double y_low = (int)last_y;
			double x_high = x_low + 1;
			double y_high = y_low + 1;

			// Compute the interpolation weights
			double x_weight = last_x - x_low;
			double y_weight = last_y - y_low;

			// TODO: Bilinear interpolation
			v_tmp(x, y) = x_weight * y_weight * v_in(x_high, y_high) +
				(1 - x_weight) * y_weight * v_in(x_low, y_high) +
				x_weight * (1 - y_weight) * v_in(x_high, y_low) +
				(1 - x_weight) * (1 - y_weight) * v_in(x_low, y_low);
		}
	}

	// Copy the values in temp to the original buffers
	u_in = u_tmp;
	v_in = v_tmp;
}

void FluidSim::MacCormackUpdate(const Array2d& d, const Array2d& d_forward, const Array2d& u,  const Array2d& u_forward, const Array2d& v, const Array2d& v_forward) {
	// move backward
	m_dt *= -1;
	advectDensitySL(u, v);
	advectVelocitySL(u, v);

	// store backward advection
	Array2d d_backward(p_density->x());
	Array2d u_backward(p_velocity->x());
	Array2d v_backward(p_velocity->y());
	m_dt *= -1;

	Array2d d_tmp(d_forward);
	Array2d u_tmp(u_forward);
	Array2d v_tmp(v_forward);
	// MacCormack Update
	for (int y = 1; y < m_res_y - 1; ++y)
		for (int x = 1; x < m_res_x - 1; ++x)
			// TODO: update d
			d_tmp(x, y) += 0.5 * (d(x, y) - d_backward(x, y));

	for (int y = 1; y < m_res_y - 1; ++y)
		for (int x = 1; x < m_res_x; ++x)
			// TODO: update u
			u_tmp(x, y) += 0.5 * (u(x, y) - u_backward(x, y));
		
	for (int y = 1; y < m_res_y; ++y)
		for (int x = 1; x < m_res_x - 1; ++x)
			// TODO: update v
			v_tmp(x, y) += 0.5 * (v(x, y) - v_backward(x, y));
	
	p_density->x() = d_tmp;
	p_velocity->x() = u_tmp;
	p_velocity->y() = v_tmp;
}

void FluidSim::MacCormackClamp(const Array2d& d, const Array2d& d_forward, const Array2d& u,  const Array2d& u_forward, const Array2d& v, const Array2d& v_forward){
	Array2d d_tmp(p_density->x());
	Array2d u_tmp(p_velocity->x());
	Array2d v_tmp(p_velocity->y());
	// Clamp density
	for (int y = 1; y < m_res_y - 1; ++y) {
		for (int x = 1; x < m_res_x - 1; ++x) {
			double last_x_velocity = (u(x, y) + u(x + 1, y)) / 2;
			double last_y_velocity = (v(x, y) + v(x, y + 1)) / 2;

			double last_x = x - m_dt * m_idx * last_x_velocity;
			double last_y = y - m_dt * m_idx * last_y_velocity;

			// Make sure the coordinates are inside the boundaries
			// Densities are known between 1 and res-2
			if (last_x < 1) last_x = 1;
			if (last_y < 1) last_y = 1;
			if (last_x > m_res_x - 2) last_x = m_res_x - 2;
			if (last_y > m_res_y - 2) last_y = m_res_y - 2;

			// Determine corners for bilinear interpolation
			int x_low = (int)last_x;
			int y_low = (int)last_y;
			int x_high = x_low + 1;
			int y_high = y_low + 1;

			double d_min = 1e10;
			double d_max = -1e10;

			d_min = min(d(x_low, y_low), d_min);
			d_min = min(d(x_low, y_high), d_min);
			d_min = min(d(x_high, y_low), d_min);
			d_min = min(d(x_high, y_high), d_min);

			d_max = max(d(x_low, y_low), d_max);
			d_max = max(d(x_low, y_high), d_max);
			d_max = max(d(x_high, y_low), d_max);
			d_max = max(d(x_high, y_high), d_max);

			// TODO: clamp d
			if (d_tmp(x, y) < d_min || d_tmp(x, y) > d_max)
				d_tmp(x, y) = d_forward(x, y);
				
		}
	}

	// Clamp velocities (u), MAC grid
	for (int y = 1; y < m_res_y - 1; ++y) {
		for (int x = 1; x < m_res_x; ++x) {
			// TODO: Compute the velocity
			double last_x_velocity = u(x, y);
			double last_y_velocity = (v(x, y) + v(x - 1, y) + v(x - 1, y + 1) + v(x, y + 1)) / 4;

			// TODO: Find the last position of the particle (in grid coordinates)
			double last_x = x - m_dt * m_idx * last_x_velocity;
			double last_y = y - m_dt * m_idx * last_y_velocity;

			// Make sure the coordinates are inside the boundaries
			// Being conservative, one can say that the velocities are known between 1.5 and res-2.5
			// (the MAC grid is inside the known densities, which are between 1 and res - 2)
			if (last_x < 1.5) last_x = 1.5;
			if (last_y < 1.5) last_y = 1.5;
			if (last_x > m_res_x - 1.5) last_x = m_res_x - 1.5;
			if (last_y > m_res_y - 2.5) last_y = m_res_y - 2.5;

			// Determine corners for bilinear interpolation
			int x_low = (int)last_x;
			int y_low = (int)last_y;
			int x_high = x_low + 1;
			int y_high = y_low + 1;

			double u_min = 1e10;
			double u_max = -1e10;

			u_min = min(u(x_low, y_low), u_min);
			u_min = min(u(x_low, y_high), u_min);
			u_min = min(u(x_high, y_low), u_min);
			u_min = min(u(x_high, y_high), u_min);

			u_max = max(u(x_low, y_low), u_max);
			u_max = max(u(x_low, y_high), u_max);
			u_max = max(u(x_high, y_low), u_max);
			u_max = max(u(x_high, y_high), u_max);

			// TODO: clamp u
			if (u_tmp(x, y) < u_min || u_tmp(x, y) > u_max)
				u_tmp(x, y) = u_forward(x, y);

		}
	}

	// Clamp velocities (v), MAC grid
	for (int y = 1; y < m_res_y; ++y) {
		for (int x = 1; x < m_res_x - 1; ++x) {
			// TODO: Compute the velocity
			double last_x_velocity = (u(x, y) + u(x + 1, y) + u(x + 1, y - 1) + u(x, y - 1)) / 4;
			double last_y_velocity = v(x, y);

			// TODO: Find the last position of the particle (in grid coordinates)
			double last_x = x - m_dt * m_idx * last_x_velocity;
			double last_y = y - m_dt * m_idx * last_y_velocity;

			// Make sure the coordinates are inside the boundaries
			// Being conservative, one can say that the velocities are known between 1.5 and res-2.5
			// (the MAC grid is inside the known densities, which are between 1 and res - 2)
			if (last_x < 1.5) last_x = 1.5;
			if (last_y < 1.5) last_y = 1.5;
			if (last_x > m_res_x - 2.5) last_x = m_res_x - 2.5;
			if (last_y > m_res_y - 1.5) last_y = m_res_y - 1.5;

			// Determine corners for bilinear interpolation
			double x_low = (int)last_x;
			double y_low = (int)last_y;
			double x_high = x_low + 1;
			double y_high = y_low + 1;
			
			double v_min = 1e10;
			double v_max = -1e10;

			v_min = min(v(x_low, y_low), v_min);
			v_min = min(v(x_low, y_high), v_min);
			v_min = min(v(x_high, y_low), v_min);
			v_min = min(v(x_high, y_high), v_min);

			v_max = max(v(x_low, y_low), v_max);
			v_max = max(v(x_low, y_high), v_max);
			v_max = max(v(x_high, y_low), v_max);
			v_max = max(v(x_high, y_high), v_max);

			// TODO: clamp v
			if (v_tmp(x, y) < v_min || v_tmp(x, y) > v_max)
				v_tmp(x, y) = v_forward(x, y);

		}
	}
	
	p_density->x() = d_tmp;
	p_velocity->x() = u_tmp;
	p_velocity->y() = v_tmp;
}